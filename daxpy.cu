
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void init(int n, double *x, double val) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = index; i < n; i += gridDim.x * blockDim.x) {
    x[i] = val;
  }
}

__global__ void daxpy(int n, double alpha, double *x, double *y) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = index; i < n; i += gridDim.x * blockDim.x) {
    y[i] = alpha * x[i] + y[i];
  }
}

int main() {
  int n = 1024;
  double *h_x, *h_y;
  hipHostAlloc(&h_x, n * sizeof(double), hipHostMallocDefault);
  hipHostAlloc(&h_y, n * sizeof(double), hipHostMallocDefault);

  double *d_x, *d_y;
  hipMalloc(&d_x, n * sizeof(double));
  hipMalloc(&d_y, n * sizeof(double));

  init<<<4, 128>>>(n, d_x, 4);
  init<<<4, 128>>>(n, d_y, 4);
  daxpy<<<4, 128>>>(n, 2, d_x, d_y);

  hipMemcpyAsync(h_x, d_x, n * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpyAsync(h_y, d_y, n * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);

  double mean = 0;
  for (int i = 0; i < n; i++) {
    mean += h_y[i] + h_x[i];
  }
  mean /= n;
  printf("mean: %f", mean);
}
